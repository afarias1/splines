#include "hip/hip_runtime.h"
#include "book.h"
#include <climits>
#include <cstdlib>
#include <iostream>
#include "timer.h"

using namespace std;

#define imin(a,b) (a<b?a:b)

const int N = 33;
const int threadsPerBlock = 256;
//blocks per grid is at most 32, but smaller if N is tiny
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);

float rand_in_range(float min, float max){
    return (max-min)*(1.0*(rand()%INT_MAX))/INT_MAX+min;
}

/* N is a global constant */
float max_cpu(float *a){
    float mval = a[0];
    for(int i=1; i<N; i++){
        if (a[i] > mval){
            mval = a[i];
        }
    }
    return mval;
}

int main( void ) {
    float   *a, *partial_result;
    float   *dev_a, *dev_result;

    GPUTimer gtime;

    //may need to change this
    int partial_size = blocksPerGrid;

    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    partial_result = (float*)malloc( partial_size*sizeof(float) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_result,
                              partial_size*sizeof(float) ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = rand_in_range(0.0f,1000.0f);
        cout << i << " " << a[i] << endl;
    }

    //initialize partial results to be all 0
    for(int i=0; i<partial_size; i++){
        partial_result[i]=0.0f;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(float),
                              hipMemcpyHostToDevice ) );


    gtime.start();

    // call the kernel
    //max_gpu_single<<<1,1>>>( dev_a, dev_result );

    gtime.stop();
    printf("Time to run kernel on GPU: %7.2f ms\n", gtime.elapsed());

    // copy the array 'result' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( partial_result, dev_result,
                              partial_size*sizeof(float),
                              hipMemcpyDeviceToHost ) );

    CPUTimer t;
    t.start();
    
    // finish up on the CPU side
    float ans = partial_result[0];
    for (int i=0; i<partial_size; i++) {
        if(partial_result[i] > ans){
            ans = partial_result[i];
        }
    }

    cout << "Max (by GPU) " << ans << endl;
    t.stop();
    printf("Time to run CPU-GPU finishing: %7.2f ms\n", 1000*t.elapsed());

    t.start();
    cout << "Max (by CPU) " << max_cpu(a) << endl;
    t.stop();
    printf("Time to run on CPU: %7.2f ms\n", 1000*t.elapsed());

    // free memory on the gpu side
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_result ) );

    // free memory on the cpu side
    free( a );
    free( partial_result );
}
